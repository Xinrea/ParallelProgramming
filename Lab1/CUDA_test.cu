//Author: Xinrea
//Date: 2018/7/5
//Basic Sample of using CUDA

#include <hip/hip_runtime.h>

#include <cstdio>

hipError_t addData(int size,int a[],int b[],int c[]);

__global__ void addKernel(int dev[],int size){
    int i = threadIdx.x;
    dev[2*size+i] = dev[i]+dev[size+i];
}

int main(int argc, char const *argv[])
{
    int size = 3;
    int a[3] = {1,2,3};
    int b[3] = {4,5,6};
    int c[3] = {0};
    hipError_t res;
    res = addData(size,a,b,c);
    if(res != hipSuccess){
        fprintf(stderr,"addData failed\n");
        return -1;
    }
    printf("%d %d %d\n",c[0],c[1],c[2]);
    res = hipDeviceReset();
    if(res != hipSuccess){
        fprintf(stderr,"hipDeviceReset failed\n");
        return -1;
    }
    return 0;
}

hipError_t addData(int size,int a[],int b[],int c[]){
    hipError_t res;
    int *dev;
    do{
        res = hipSetDevice(0);
        if (res != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed\n");
            break;
        }
        res = hipMalloc((void**)&dev,3*size*sizeof(int));
        if(res != hipSuccess){
            fprintf(stderr,"hipMalloc failed\n");
            break;
        }
        res = hipMemcpy(dev,a,size*sizeof(int),hipMemcpyHostToDevice);
        if(res != hipSuccess){
            fprintf(stderr,"hipMemcpy a failed\n");
            break;
        }
        res = hipMemcpy(dev+size,b,size*sizeof(int),hipMemcpyHostToDevice);
        if(res != hipSuccess){
            fprintf(stderr,"hipMemcpy b failed\n");
            break;
        }
        addKernel<<<1,size>>>(dev,size);
        res = hipGetLastError();
        if(res != hipSuccess){
            fprintf(stderr,"addData failed\n");
            break;
        }
        hipDeviceSynchronize();
        res = hipMemcpy(c,dev+2*size,size*sizeof(int),hipMemcpyDeviceToHost);
        if(res != hipSuccess){
            fprintf(stderr,"hipMemcpy c failed\n");
            break;
        }
    } while(0);
    hipFree(dev);
    return res;
}
